#include "hip/hip_runtime.h"
#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


struct Node
{
int starting;
int no_of_edges;
};

extern "C" __global__ void BFSGraph_kernel0(int lfpriv__no_of_nodes, _Bool * h_graph_mask, _Bool * h_graph_visited, _Bool * h_updating_graph_mask)
{
unsigned int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
#pragma acc  parallel loop num_workers(64) gang worker independent present(h_graph_mask[0:no_of_nodes], h_graph_visited[0:no_of_nodes], h_updating_graph_mask[0:no_of_nodes]) private(i) firstprivate(no_of_nodes) num_gangs(((int)ceil((((float)no_of_nodes)/64.0F))))
if (lwpriv__i<lfpriv__no_of_nodes)
{
h_updating_graph_mask[lwpriv__i]=0;
h_graph_mask[lwpriv__i]=0;
h_graph_visited[lwpriv__i]=0;
}
}

extern "C" __global__ void BFSGraph_kernel1(_Bool * h_graph_mask, _Bool * h_graph_visited, int source)
{
/* set the source node as true in the mask */
h_graph_mask[source]=1;
h_graph_visited[source]=1;
}

extern "C" __global__ void BFSGraph_kernel2(int lfpriv__no_of_nodes, int lfpriv__source, int * h_cost)
{
unsigned int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
#pragma acc  parallel loop num_workers(64) gang worker independent present(h_cost[0:no_of_nodes]) private(i) firstprivate(no_of_nodes, source) num_gangs(((int)ceil((((float)no_of_nodes)/64.0F))))
if (lwpriv__i<lfpriv__no_of_nodes)
{
h_cost[lwpriv__i]=( - 1);
if (lwpriv__i==lfpriv__source)
{
h_cost[lfpriv__source]=0;
}
}
}

extern "C" __global__ void BFSGraph_kernel3(int lfpriv__no_of_nodes, int * h_cost, int * h_graph_edges, _Bool * h_graph_mask, struct Node * h_graph_nodes, _Bool * h_graph_visited, _Bool * h_updating_graph_mask)
{
int lwpriv__tid;
lwpriv__tid=(threadIdx.x+(blockIdx.x*64));
#pragma acc  parallel loop num_workers(64) gang worker independent present(h_cost[0:no_of_nodes], h_graph_edges[0:edge_list_size], h_graph_mask[0:no_of_nodes], h_graph_nodes[0:no_of_nodes], h_graph_visited[0:no_of_nodes], h_updating_graph_mask[0:no_of_nodes]) private(tid) firstprivate(no_of_nodes) num_gangs(((int)ceil((((float)no_of_nodes)/64.0F))))
if (lwpriv__tid<lfpriv__no_of_nodes)
{
if (h_graph_mask[lwpriv__tid]==1)
{
int i;
h_graph_mask[lwpriv__tid]=0;
for (i=h_graph_nodes[lwpriv__tid].starting; i<(h_graph_nodes[lwpriv__tid].no_of_edges+h_graph_nodes[lwpriv__tid].starting); i ++ )
{
int id;
id=h_graph_edges[i];
if ( ! h_graph_visited[id])
{
h_cost[id]=(h_cost[lwpriv__tid]+1);
h_updating_graph_mask[id]=1;
}
}
}
}
}

extern "C" __global__ void BFSGraph_kernel4(_Bool * restrict lgred__stop, int lfpriv__no_of_nodes, _Bool * h_graph_mask, _Bool * h_graph_visited, _Bool * h_updating_graph_mask)
{
int lwpriv__tid;
int _bid;
int _bsize;
int _tid;
volatile _Bool __shared__ lwreds__stop[64];
int _ti_100_1001;
int _ti_100_1002;
int _ti_100_1003;
_tid=((threadIdx.x+(threadIdx.y*blockDim.x))+(threadIdx.z*(blockDim.x*blockDim.y)));
_bsize=((blockDim.x*blockDim.y)*blockDim.z);
_bid=((blockIdx.x+(blockIdx.y*gridDim.x))+(blockIdx.z*(gridDim.x*gridDim.y)));
lwreds__stop[_tid]=0;
lwpriv__tid=(threadIdx.x+(blockIdx.x*64));
#pragma acc  parallel loop num_workers(64) gang worker vector reduction(||: stop) present(h_graph_mask[0:no_of_nodes], h_graph_visited[0:no_of_nodes], h_updating_graph_mask[0:no_of_nodes]) private(tid) firstprivate(no_of_nodes) num_gangs(((int)ceil((((float)no_of_nodes)/64.0F))))
if (lwpriv__tid<lfpriv__no_of_nodes)
{
if (h_updating_graph_mask[lwpriv__tid]==1)
{
h_graph_mask[lwpriv__tid]=1;
h_graph_visited[lwpriv__tid]=1;
lwreds__stop[_tid]=1;
h_updating_graph_mask[lwpriv__tid]=0;
}
}
__syncthreads();
_ti_100_1002=_bsize;
for (_ti_100_1001=(_bsize>>1); _ti_100_1001>0; _ti_100_1001>>=1)
{
if (_tid<_ti_100_1001)
{
lwreds__stop[_tid]=(lwreds__stop[_tid]||lwreds__stop[(_tid+_ti_100_1001)]);
}
_ti_100_1003=(_ti_100_1002&1);
if (_ti_100_1003==1)
{
if (_tid==0)
{
lwreds__stop[_tid]=(lwreds__stop[_tid]||lwreds__stop[(_tid+(_ti_100_1002-1))]);
}
}
_ti_100_1002=_ti_100_1001;
if (_ti_100_1001>32)
{
__syncthreads();
}
}
if (_tid==0)
{
lgred__stop[_bid]=lwreds__stop[_tid];
}
}

